#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

void query_device() {

	int deviceCount { 0 };
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0) {
		std::cout << "No CUDA support device found" << std::endl;
	}

	int devNo { 0 };
	hipDeviceProp_t iProp;
	hipGetDeviceProperties(&iProp, devNo);

	std::cout << "Device " << devNo << ": " << iProp.name << std::endl;
	std::cout << "  Number of multiprocessors: " << iProp.multiProcessorCount
			<< std::endl;
	std::cout << "  clock rate: " << iProp.clockRate << std::endl;
	std::cout << "  Compute capability: " << iProp.major << "." << iProp.minor
			<< std::endl;
	std::cout << "  Total amount of global memory: "
			<< iProp.totalGlobalMem / 1024 << " KB" << std::endl;
	std::cout << "  Total amount of constant memory: "
			<< iProp.totalConstMem / 1024 << " KB" << std::endl;
	std::cout << "  Total amount of shared memory per block: "
			<< iProp.sharedMemPerBlock / 1024 << " KB" << std::endl;
	std::cout << "  Total number of registers available per block: "
			<< iProp.regsPerBlock << std::endl;
	std::cout << "  Warp size: " << iProp.warpSize << std::endl;
	std::cout << "  Maximum number of threads per block: "
			<< iProp.maxThreadsPerBlock << std::endl;
	std::cout << "  Maximum Grid size: (" << iProp.maxGridSize[0] << ", "
			<< iProp.maxGridSize[1] << ", " << iProp.maxGridSize[2] << ")"
			<< std::endl;
	std::cout << "  Maximum block dimension: (" << iProp.maxThreadsDim[0]
			<< ", " << iProp.maxThreadsDim[1] << ", " << iProp.maxThreadsDim[2]
			<< ")" << std::endl;
}

int main() {
	query_device();
	return 0;
}
